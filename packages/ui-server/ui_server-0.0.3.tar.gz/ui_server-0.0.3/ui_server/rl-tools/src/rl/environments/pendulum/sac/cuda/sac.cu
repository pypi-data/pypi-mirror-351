#include "hip/hip_runtime.h"
#ifdef RL_TOOLS_DEBUG
#define RL_TOOLS_DEBUG_DEVICE_CUDA_SYNCHRONIZE_STATUS_CHECK
#endif
#define RL_TOOLS_OPERATIONS_CPU_MUX_INCLUDE_CUDA
#include <rl_tools/operations/cpu_mux.h>
#include <rl_tools/nn/optimizers/adam/instance/operations_cuda.h>
#include <rl_tools/nn/operations_cpu_mux.h>
#include <rl_tools/nn/layers/sample_and_squash/operations_cuda.h>
#include <rl_tools/rl/environments/pendulum/operations_cpu.h>
#include <rl_tools/nn_models/mlp/operations_generic.h>
#include <rl_tools/nn_models/sequential/operations_generic.h>

#include <rl_tools/nn/optimizers/adam/operations_generic.h>
#include <rl_tools/rl/algorithms/sac/operations_cuda.h>

#include <rl_tools/rl/algorithms/sac/loop/core/config.h>
#include <rl_tools/rl/loop/steps/evaluation/config.h>
#include <rl_tools/rl/loop/steps/timing/config.h>
#include <rl_tools/rl/algorithms/sac/loop/core/operations_generic.h>
#include <rl_tools/rl/loop/steps/evaluation/operations_generic.h>
#include <rl_tools/rl/loop/steps/timing/operations_cpu.h>

#include <rl_tools/rl/components/off_policy_runner/operations_cuda.h>

namespace rlt = rl_tools;

using DEVICE = rlt::devices::DEVICE_FACTORY_CUDA<>;
#ifndef _MSC_VER
using DEVICE_INIT = rlt::devices::DEVICE_FACTORY<>;
#else
using DEVICE_INIT = rlt::devices::DefaultCPU; // for some reason MKL makes problems in this case (this example seems cursed)
#endif
using RNG = decltype(rlt::random::default_engine(typename DEVICE::SPEC::RANDOM{}));
using RNG_INIT = decltype(rlt::random::default_engine(typename DEVICE_INIT::SPEC::RANDOM{}));
using T = float;
using TI = typename DEVICE::index_t;

using PENDULUM_SPEC = rlt::rl::environments::pendulum::Specification<T, TI, rlt::rl::environments::pendulum::DefaultParameters<T>>;
using ENVIRONMENT = rlt::rl::environments::Pendulum<PENDULUM_SPEC>;
struct LOOP_CORE_PARAMETERS: rlt::rl::algorithms::sac::loop::core::DefaultParameters<T, TI, ENVIRONMENT>{
    struct SAC_PARAMETERS: rlt::rl::algorithms::sac::DefaultParameters<T, TI, ENVIRONMENT::ACTION_DIM>{
        static constexpr TI ACTOR_BATCH_SIZE = 100;
        static constexpr TI CRITIC_BATCH_SIZE = 100;
    };
    static constexpr TI STEP_LIMIT = 20000;
    static constexpr TI ACTOR_NUM_LAYERS = 3;
    static constexpr TI ACTOR_HIDDEN_DIM = 64;
    static constexpr TI CRITIC_NUM_LAYERS = 3;
    static constexpr TI CRITIC_HIDDEN_DIM = 64;
    static constexpr bool COLLECT_EPISODE_STATS = false;
    static constexpr TI EPISODE_STATS_BUFFER_SIZE = 0;
};
template <typename RNG>
using LOOP_CORE_CONFIG = rlt::rl::algorithms::sac::loop::core::Config<T, TI, RNG, ENVIRONMENT, LOOP_CORE_PARAMETERS>;

struct LOOP_EVAL_PARAMETERS: rlt::rl::loop::steps::evaluation::Parameters<T, TI, LOOP_CORE_CONFIG<RNG>>{
    static constexpr TI NUM_EVALUATION_EPISODES = 100;
};
template <typename RNG>
using LOOP_EVAL_CONFIG = rlt::rl::loop::steps::evaluation::Config<LOOP_CORE_CONFIG<RNG>, LOOP_EVAL_PARAMETERS>;
template <typename RNG>
using LOOP_TIMING_CONFIG = rlt::rl::loop::steps::timing::Config<LOOP_EVAL_CONFIG<RNG>>;
template <typename RNG>
using LOOP_CONFIG = LOOP_TIMING_CONFIG<RNG>;

using LOOP_STATE = LOOP_CONFIG<RNG>::template State<LOOP_CONFIG<RNG>>;
using LOOP_STATE_INIT = LOOP_CONFIG<RNG_INIT>::template State<LOOP_CONFIG<RNG_INIT>>;


int main(){
    DEVICE device;
    DEVICE_INIT device_init;
    LOOP_STATE ts;
    LOOP_STATE_INIT ts_init;
    using CONFIG = decltype(ts)::CONFIG;
    using CORE_PARAMETERS = CONFIG::CORE_PARAMETERS;
    using EVAL_PARAMETERS = CONFIG::EVALUATION_PARAMETERS;
    rlt::init(device);
    rlt::malloc(device, ts);
    rlt::malloc(device_init, ts_init);
    rlt::init(device_init, ts_init, 1);
    rlt::copy(device_init, device, ts_init, ts);
//    rlt::copy(device_init, device, ts_init.off_policy_runner, ts.off_policy_runner);

#ifdef _MSC_VER
    CONFIG::ENVIRONMENT env_eval;
    RNG_INIT rng_eval;
    rlt::rl::environments::DummyUI ui;
#endif

    decltype(ts.off_policy_runner)* off_policy_runner_pointer;
    hipMalloc(&off_policy_runner_pointer, sizeof(decltype(ts.off_policy_runner)));
    hipMemcpy(off_policy_runner_pointer, &ts.off_policy_runner, sizeof(decltype(ts.off_policy_runner)), hipMemcpyHostToDevice);
    rlt::check_status(device);

    TI step = 0;
    ts.rng = rlt::random::next(device.random, ts.rng);
    bool finished = false;
    auto start_time = std::chrono::high_resolution_clock::now();
    while(!finished){
        rlt::set_step(device, device.logger, step);
        rlt::step(device, ts.off_policy_runner, off_policy_runner_pointer, ts.actor_critic.actor, ts.actor_buffers_eval, ts.rng);
        if(step > CONFIG::CORE_PARAMETERS::N_WARMUP_STEPS){
            if(step % CONFIG::CORE_PARAMETERS::SAC_PARAMETERS::CRITIC_TRAINING_INTERVAL == 0) {
                hipStream_t critic_training_streams[2];
                for(int critic_i = 0; critic_i < 2; critic_i++){
                    hipStreamCreate(&critic_training_streams[critic_i]);
//                    device.stream = critic_training_streams[critic_i]; // parallel streams actually make it slightly worse (bandwidth bound?)
                    rlt::gather_batch(device, off_policy_runner_pointer, ts.critic_batch, ts.rng);
                    rlt::randn(device, ts.action_noise_critic, ts.rng);
                    rlt::train_critic(device, ts.actor_critic, critic_i == 0 ? ts.actor_critic.critic_1 : ts.actor_critic.critic_2, ts.critic_batch, ts.critic_optimizers[critic_i], ts.actor_buffers[critic_i], ts.critic_buffers[critic_i], ts.critic_training_buffers[critic_i], ts.action_noise_critic, ts.rng);
                }
                for(int critic_i = 0; critic_i < 2; critic_i++){
                    hipStreamSynchronize(critic_training_streams[critic_i]);
                    hipStreamDestroy(critic_training_streams[critic_i]);
                }
                device.stream = 0;
            }
            if(step % CONFIG::CORE_PARAMETERS::SAC_PARAMETERS::ACTOR_TRAINING_INTERVAL == 0) {
                {
                    rlt::gather_batch(device, off_policy_runner_pointer, ts.actor_batch, ts.rng);
                    rlt::randn(device, ts.action_noise_actor, ts.rng);
                    rlt::train_actor(device, ts.actor_critic, ts.actor_batch, ts.actor_optimizer, ts.actor_buffers[0], ts.critic_buffers[0], ts.actor_training_buffers, ts.action_noise_actor, ts.rng);
                }
                rlt::update_critic_targets(device, ts.actor_critic);
            }
        }
#ifndef BENCHMARK
        if(step % 1000 == 0){
            rlt::copy(device, device_init, ts.actor_critic.actor, ts_init.actor_critic.actor);
#ifdef _MSC_VER
            using RESULT_SPEC = rlt::rl::utils::evaluation::Specification<T, TI, typename LOOP_STATE::CONFIG::ENVIRONMENT_EVALUATION, EVAL_PARAMETERS::NUM_EVALUATION_EPISODES, CORE_PARAMETERS::EPISODE_STEP_LIMIT>;
            rlt::rl::utils::evaluation::Result<RESULT_SPEC> result;
            rlt::evaluate(device_init, env_eval, ui, ts_init.actor_critic.actor, result, ts_init.actor_deterministic_evaluation_buffers, rng_eval, false);
//            auto result = rlt::evaluate(device_init, env_eval, ui, ts_init.actor_critic.actor, rlt::rl::utils::evaluation::Specification<EVAL_PARAMETERS::NUM_EVALUATION_EPISODES, CORE_PARAMETERS::EPISODE_STEP_LIMIT>(), ts_init.actor_deterministic_evaluation_buffers, rng_eval, false);
#else
            using RESULT_SPEC = rlt::rl::utils::evaluation::Specification<T, TI, typename LOOP_STATE::CONFIG::ENVIRONMENT_EVALUATION, EVAL_PARAMETERS::NUM_EVALUATION_EPISODES, CORE_PARAMETERS::EPISODE_STEP_LIMIT>;
            rlt::rl::utils::evaluation::Result<RESULT_SPEC> result;
            rlt::evaluate(device_init, ts_init.env_eval, ts_init.ui, ts_init.actor_critic.actor, result, ts_init.actor_deterministic_evaluation_buffers, ts_init.rng_eval, rlt::Mode<rlt::mode::Evaluation<>>{});
#endif
            rlt::log(device_init, device_init.logger, "Step: ", step, " Mean return: ", result.returns_mean);
//            add_scalar(device, device.logger, "evaluation/return/mean", result.returns_mean);
//            add_scalar(device, device.logger, "evaluation/return/std", result.returns_std);
        }
#endif
        step++;
        finished = step > CORE_PARAMETERS::STEP_LIMIT;
    }
    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed_seconds = end_time - start_time;
    std::cout << "total time: " << elapsed_seconds.count() << "s" << std::endl;
    rlt::free(device, ts);
    rlt::free(device_init, ts_init);
}

// benchmark training should take < 2s on P1

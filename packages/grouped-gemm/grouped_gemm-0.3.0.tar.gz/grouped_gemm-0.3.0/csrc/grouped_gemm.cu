#include "grouped_gemm.h"
#include "fill_arguments.cuh"

#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/KernelUtils.h>
#include <c10/util/BFloat16.h>
#include <c10/cuda/CUDAStream.h>
#include <hipcub/hipcub.hpp>
#include <torch/extension.h>

#include "cutlass/bfloat16.h"
#include "cutlass/complex.h"
#include "cutlass/gemm/kernel/gemm_grouped.h"
#include "cutlass/gemm/kernel/default_gemm_grouped.h"
#include "cutlass/gemm/device/gemm_grouped.h"

#include <type_traits>

namespace grouped_gemm {

#define CUDA_CALL(code)					    \
  do {                                                      \
    hipError_t status = code;                              \
    std::string err = hipGetErrorString(status);           \
    TORCH_CHECK(status == hipSuccess, err);		    \
  } while (0)

#define CUBLAS_CALL(code)					  \
  do {								  \
    hipblasStatus_t status = code;				  \
    TORCH_CHECK(status == HIPBLAS_STATUS_SUCCESS, "CuBLAS Error"); \
  } while (0)

#define GROUPED_GEMM_STRINGIFY_HELPER(x) #x
#define GROUPED_GEMM_STRINGIFY(x) \
  GROUPED_GEMM_STRINGIFY_HELPER(x)

template <bool trans>
using GroupedGemmInputLayout = std::conditional_t<trans, ::cutlass::layout::ColumnMajor, ::cutlass::layout::RowMajor>;

using GroupedGemmConfig = ::cutlass::gemm::device::DefaultGemmConfiguration<
  ::cutlass::arch::OpClassTensorOp,
  ::cutlass::arch::Sm80,
  ::cutlass::bfloat16_t,
  ::cutlass::bfloat16_t,
  ::cutlass::bfloat16_t,
  float
>;

// TODO(tgale): Update this for SM90 when it's supported by CUTLASS.
template <bool trans_a, bool trans_b>
using GroupedGemmKernel = typename cutlass::gemm::kernel::DefaultGemmGrouped<
  // A operand.
  ::cutlass::bfloat16_t,
  GroupedGemmInputLayout<trans_a>,
  ::cutlass::ComplexTransform::kNone,
  GroupedGemmConfig::kAlignmentA,
  // B operand.
  ::cutlass::bfloat16_t,
  GroupedGemmInputLayout<trans_b>,
  ::cutlass::ComplexTransform::kNone,
  GroupedGemmConfig::kAlignmentB,
  // C operand.
  ::cutlass::bfloat16_t,
  ::cutlass::layout::RowMajor,
  float,
  ::cutlass::arch::OpClassTensorOp,
  ::cutlass::arch::Sm80,
  GroupedGemmConfig::ThreadblockShape,
  GroupedGemmConfig::WarpShape,
  GroupedGemmConfig::InstructionShape,
  GroupedGemmConfig::EpilogueOutputOp,
  // NOTE: Threadblock swizzling is currently not supported by CUTLASS's grouped kernels.
  // This parameter is passed in at present to match the APIs of other kernels. The parameter
  // is unused within the kernel.
  ::cutlass::gemm::threadblock::GemmBatchedIdentityThreadblockSwizzle,
  // TODO(tgale): Tune this for SM90.
  GroupedGemmConfig::kStages>::GemmKernel;

template <bool trans_a, bool trans_b>
using GemmGrouped = ::cutlass::gemm::device::GemmGrouped<GroupedGemmKernel<trans_a, trans_b>>;

template <typename T>
torch::Tensor CopyToDevice(const std::vector<T> &x, const torch::Device &device) {
  size_t bytes = x.size() * sizeof(T);
  auto options = torch::TensorOptions().dtype(torch::kInt8).device(device);
  torch::Tensor out = torch::empty(bytes, options);

  CUDA_CALL(hipMemcpyAsync(out.data_ptr(),
			    x.data(), bytes,
			    hipMemcpyHostToDevice,
			    c10::cuda::getCurrentCUDAStream()));
  return out;
}

template <typename T>
static void ReorderArray(T* data, const std::vector<size_t>& indices) {
    // For now, simply create a copy of the data and then copy over to the original.
    std::vector<T> copy(data, data + indices.size());
    for (size_t i = 0; i < indices.size(); ++i) {
        data[i] = copy.at(indices[i]);
    }
}

template <typename T>
torch::Tensor TypedEmpty(size_t numel, const torch::Device& device) {
    return torch::empty(numel * sizeof(T), torch::dtype(torch::kInt8).device(device));
}

struct RawGemmArguments {
  torch::Tensor lda, ldb, ldc, ptr_a, ptr_b, ptr_c, problem_sizes;
  int threadblock_count{};
};

template <
  typename Gemm,
  typename ElementA, typename ElementB, typename ElementC
>
RawGemmArguments MakeArgumentsOnDevice(int num_experts, const torch::Device& device) {
    TORCH_CHECK(
        num_experts <= kMaxExperts,
        "At most ", kMaxExperts,
        " experts are supported when batch_sizes is a CUDA tensor, but got ", num_experts
    );

    return RawGemmArguments {
      .lda = TypedEmpty<int64_t>(num_experts, device),
      .ldb = TypedEmpty<int64_t>(num_experts, device),
      .ldc = TypedEmpty<int64_t>(num_experts, device),
      .ptr_a = TypedEmpty<ElementA*>(num_experts, device),
      .ptr_b = TypedEmpty<ElementB*>(num_experts, device),
      .ptr_c = TypedEmpty<ElementC*>(num_experts, device),
      .problem_sizes = TypedEmpty<cutlass::gemm::GemmCoord>(num_experts, device),

      // We don't know the problem dimensions on the host, so we just base the number of threadblocks on occupancy here.
      .threadblock_count = Gemm::sufficient(),
    };
}

template <
  bool kDynamicK,
  typename Gemm,
  typename ElementA, typename ElementB, typename ElementC,
  typename LayoutA, typename LayoutB, typename LayoutC
>
RawGemmArguments MakeArgumentsOnHost(torch::Tensor a,
				     torch::Tensor b,
				     torch::Tensor c,
				     torch::Tensor batch_sizes,
				     ::cutlass::gemm::GemmCoord coord_template,
				     int64_t num_experts) {
  std::vector<::cutlass::gemm::GemmCoord> problem_sizes_host(num_experts);

  // Create the host arrays of leading dimension data and pointer data.
  std::vector<int64_t> lda_host(num_experts), ldb_host(num_experts), ldc_host(num_experts);
  int64_t elements_a = 0, elements_b = 0, elements_c = 0;

  std::vector<ElementA *> ptr_a_host(num_experts), ptr_b_host(num_experts), ptr_c_host(num_experts);

  for (int i = 0; i < num_experts; ++i) {
    auto& problem = problem_sizes_host[i];
    problem = coord_template;
    (kDynamicK ? problem.k() : problem.m()) = batch_sizes.data_ptr<int64_t>()[i];

    lda_host[i] = LayoutA::packed({problem.m(), problem.k()}).stride(0);
    ldb_host[i] = LayoutB::packed({problem.k(), problem.n()}).stride(0);
    ldc_host[i] = LayoutC::packed({problem.m(), problem.n()}).stride(0);

    ptr_a_host[i] = (ElementA*)a.data_ptr() + elements_a;
    ptr_b_host[i] = (ElementB*)b.data_ptr() + elements_b;
    ptr_c_host[i] = (ElementC*)c.data_ptr() + elements_c;

    elements_a += problem.m() * problem.k();
    elements_b += problem.k() * problem.n();
    elements_c += problem.m() * problem.n();

    if (problem.k() == 0) {
      // CUTLASS doesn't handle problems with `k=0` correctly, see https://github.com/NVIDIA/cutlass/pull/1593.
      // Until a fix is available on the CUTLASS side, handle these problems by ourselves:
      //   * set the output to zero with `hipMemsetAsync()`
      //   * make this problem a no-op by setting `m=0` and `n=0` (CUTLASS can handle the outer dimensions being zero)
      CUDA_CALL(hipMemsetAsync(ptr_c_host[i],
        0,
        problem.m() * problem.n() * sizeof(ElementC),
        c10::cuda::getCurrentCUDAStream()));

      problem.m() = 0;
      problem.n() = 0;
    }
  }

  // Only sort problems when K are different
  if (kDynamicK) {
      std::vector<size_t> indices(num_experts);
      std::iota(indices.begin(), indices.end(), 0);
      std::stable_sort(indices.begin(), indices.end(), [&problem_sizes_host](size_t i, size_t j) {
          return problem_sizes_host[i].k() > problem_sizes_host[j].k();
      });

      ReorderArray(problem_sizes_host.data(), indices);
      ReorderArray(lda_host.data(), indices);
      ReorderArray(ldb_host.data(), indices);
      ReorderArray(ldc_host.data(), indices);
      ReorderArray(ptr_a_host.data(), indices);
      ReorderArray(ptr_b_host.data(), indices);
      ReorderArray(ptr_c_host.data(), indices);
  }

  // Copy the problem sizes, pointers and leading dimension data to the device.
  return RawGemmArguments {
    .lda = CopyToDevice(lda_host, a.device()),
    .ldb = CopyToDevice(ldb_host, a.device()),
    .ldc = CopyToDevice(ldc_host, a.device()),
    .ptr_a = CopyToDevice(ptr_a_host, a.device()),
    .ptr_b = CopyToDevice(ptr_b_host, a.device()),
    .ptr_c = CopyToDevice(ptr_c_host, a.device()),
    .problem_sizes = CopyToDevice(problem_sizes_host, a.device()),

    // We know the problem dimensions on the host, so we can calculate the number of threadblocks based on that.
    .threadblock_count = Gemm::sufficient(problem_sizes_host.data(), num_experts),
  };
}

template <
  bool kDynamicK,
  typename Gemm,
  typename ElementA, typename ElementB, typename ElementC,
  typename LayoutA, typename LayoutB, typename LayoutC
>
typename Gemm::Arguments MakeArguments(torch::Tensor a,
				       torch::Tensor b,
				       torch::Tensor c,
				       torch::Tensor batch_sizes,
				       ::cutlass::gemm::GemmCoord coord_template,
				       int64_t num_experts) {
  RawGemmArguments raw_args;
  if (batch_sizes.is_cuda()) {
    raw_args = MakeArgumentsOnDevice<
      Gemm, ElementA, ElementB, ElementC
    >(num_experts, a.device());
  } else {
    raw_args = MakeArgumentsOnHost<
      kDynamicK,
      Gemm,
      ElementA, ElementB, ElementC,
      LayoutA, LayoutB, LayoutC
    >(a, b, c, batch_sizes, coord_template, num_experts);
  }

  // Validate the result.
  if (!raw_args.threadblock_count) {
    TORCH_CHECK(false, "Grouped GEMM execution not possible with HW");
  }

  typename Gemm::EpilogueOutputOp::Params epilogue_op(/*alpha=*/1.0f, /*beta=*/0.0f);
  // We currently always use `GroupScheduleMode::kDeviceOnly`, which doesn't use `host_problem_sizes` at all,
  // so we can safely pass `nullptr` for `host_problem_sizes`.
  // TODO(tgale): Experiment with `GroupScheduleMode::kHostPrecompute` for `batch_sizes.is_cpu()`, where we
  // know the problem dimensions on the host.
  typename Gemm::Arguments arguments((cutlass::gemm::GemmCoord*)raw_args.problem_sizes.data_ptr(),
				     (int)num_experts,
				     (int)raw_args.threadblock_count,
				     epilogue_op,
				     (ElementA**)raw_args.ptr_a.data_ptr(),
				     (ElementB**)raw_args.ptr_b.data_ptr(),
				     (ElementC**)raw_args.ptr_c.data_ptr(),
				     (ElementC**)raw_args.ptr_c.data_ptr(),
				     /*lda=*/(int64_t*)raw_args.lda.data_ptr(),
				     /*ldb=*/(int64_t*)raw_args.ldb.data_ptr(),
				     /*ldc=*/(int64_t*)raw_args.ldc.data_ptr(),
				     /*ldd=*/(int64_t*)raw_args.ldc.data_ptr(),
				     /*host_problem_sizes=*/nullptr);
  return arguments;
}

template <
  bool trans_a,
  typename ElementA, typename ElementB, typename ElementC,
  typename LayoutA, typename LayoutB, typename LayoutC,
  typename Arguments
>
void FillCutlassArguments(int num_experts,
			  torch::Tensor batch_sizes,
			  torch::Tensor a,
			  torch::Tensor b,
			  torch::Tensor c,
			  const Arguments& arguments,
			  ::cutlass::gemm::GemmCoord coord_template) {
  // Convert the batch sizes to the format CUTLASS understands on the device.
  // Use a single block here because:
  //   * the number of elements to process is microscopically small
  //   * we don't need any additional global memory
  FillArguments<
      /*kDynamicK*/trans_a,
      ElementA, ElementB, ElementC,
      LayoutA, LayoutB, LayoutC
  ><<<1, kMaxExperts, 0, c10::cuda::getCurrentCUDAStream()>>>(
      num_experts, batch_sizes.data_ptr<int64_t>(),
      (ElementA*)a.data_ptr(), (ElementB*)b.data_ptr(), (ElementC*)c.data_ptr(),
      arguments, coord_template
  );
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

template <typename Args>
void RemoveK0Problems(int num_experts, const Args& arguments) {
  // For zeroing out the outputs (which might be arbitrarily large), we want to use
  // as many threadblocks as possible in order to hit the maximum possible global memory bandwidth.
  // `arguments.threadblock_count`, which we will use for the grouped GEMM proper,
  // should be a good approximation for this.
  // When the `k=0` case is fixed in CUTLASS, we can completely remove this function.
  ZeroOutK0Outputs<><<<
    arguments.threadblock_count, at::cuda::detail::CUDA_NUM_THREADS, 0, c10::cuda::getCurrentCUDAStream()
  >>>(
    num_experts, arguments
  );
  IgnoreK0Problems<><<<
    1, kMaxExperts, 0, c10::cuda::getCurrentCUDAStream()
  >>>(
    num_experts, arguments
  );
}

template <bool trans_a, bool trans_b>
torch::Tensor CutlassGroupedGemm(torch::Tensor a,
				 torch::Tensor b,
				 torch::Tensor c,
				 torch::Tensor batch_sizes,
				 ::cutlass::gemm::GemmCoord coord_template) {
  using Gemm = GemmGrouped<trans_a, trans_b>;
  using LayoutA = typename Gemm::LayoutA;
  using LayoutB = typename Gemm::LayoutB;
  using LayoutC = typename Gemm::LayoutC;

  using ElementA = typename Gemm::ElementA;
  using ElementB = typename Gemm::ElementB;
  using ElementC = typename Gemm::ElementC;

  Gemm gemm;
  int64_t num_experts = batch_sizes.size(0);
  auto arguments = MakeArguments<
    /*kDynamicK*/trans_a,
    Gemm,
    ElementA, ElementB, ElementC,
    LayoutA, LayoutB, LayoutC
  >(a, b, c, batch_sizes, coord_template, num_experts);
  int64_t workspace_size = gemm.get_workspace_size(arguments);
  auto options = torch::TensorOptions().dtype(torch::kInt8).device(a.device());
  torch::Tensor workspace = torch::empty(workspace_size, options);

  if (batch_sizes.is_cuda()) {
      FillCutlassArguments<
        trans_a,
        ElementA, ElementB, ElementC,
        LayoutA, LayoutB, LayoutC
      >(num_experts, batch_sizes, a, b, c, arguments, coord_template);

      RemoveK0Problems<>(num_experts, arguments);
  }

  // Initialize the kernel.
  if(gemm.initialize(arguments, workspace.data_ptr()) != cutlass::Status::kSuccess) {
    TORCH_CHECK(false, "Failed to initialize CUTLASS Grouped GEMM");
  }

  // Execute the kernel in the current stream.
  if(gemm.run(c10::cuda::getCurrentCUDAStream()) != cutlass::Status::kSuccess) {
    TORCH_CHECK(false, "Failed to run CUTLASS Grouped GEMM");
  }
  return c;
}

void CublasGemm(c10::BFloat16 *a, int64_t a_rows, int64_t a_cols, bool trans_a,
		c10::BFloat16 *b, int64_t b_rows, int64_t b_cols, bool trans_b,
		c10::BFloat16 *c, int64_t c_rows, int64_t c_cols) {
  int m = trans_b ? b_rows : b_cols;
  int k = trans_b ? b_cols : b_rows;
  int n = trans_a ? a_cols : a_rows;

  int lda = trans_a ? n : k;
  int ldb = trans_b ? k : m;
  hipblasOperation_t transpose_a = trans_a ? HIPBLAS_OP_T : HIPBLAS_OP_N;
  hipblasOperation_t transpose_b = trans_b ? HIPBLAS_OP_T : HIPBLAS_OP_N;

  float alpha = 1.0, beta = 0.0;
  CUBLAS_CALL(hipblasGemmEx(at::cuda::getCurrentCUDABlasHandle(),
			   transpose_b, transpose_a,
			   m, n, k, &alpha,
			   b, HIP_R_16BF, ldb,
			   a, HIP_R_16BF, lda,
			   &beta,
			   c, HIP_R_16BF, c_cols, HIP_R_32F,
			   HIPBLAS_GEMM_DEFAULT));
}

void CublasGroupedGemm(torch::Tensor a,
		       torch::Tensor b,
		       torch::Tensor c,
		       torch::Tensor batch_sizes,
		       bool trans_b) {
  int64_t bs = batch_sizes.size(0), k = a.size(1);
  int64_t n = trans_b ? b.size(1) : b.size(2);
  int64_t b_rows = b.size(1), b_cols = b.size(2);
  c10::BFloat16* a_ptr = a.data_ptr<c10::BFloat16>();
  c10::BFloat16* b_ptr = b.data_ptr<c10::BFloat16>();
  c10::BFloat16* c_ptr = c.data_ptr<c10::BFloat16>();
  for (int i = 0; i < bs; ++i) {
    int64_t m = batch_sizes.data_ptr<int64_t>()[i];
    CublasGemm(a_ptr, m, k, /*trans_a=*/false,
	       b_ptr, b_rows, b_cols, trans_b,
	       c_ptr, m, n);
    a_ptr += m * k;
    b_ptr += b_rows * b_cols;
    c_ptr += m * n;
  }
}

void CublasGroupedGemmVariableK(torch::Tensor a,
				torch::Tensor b,
				torch::Tensor c,
				torch::Tensor batch_sizes) {
  int64_t bs = batch_sizes.size(0), m = a.size(1), n = b.size(1);
  c10::BFloat16* a_ptr = a.data_ptr<c10::BFloat16>();
  c10::BFloat16* b_ptr = b.data_ptr<c10::BFloat16>();
  c10::BFloat16* c_ptr = c.data_ptr<c10::BFloat16>();
  for (int i = 0; i < bs; ++i) {
    int64_t k = batch_sizes.data_ptr<int64_t>()[i];
    CublasGemm(a_ptr, k, m, /*trans_a=*/true,
	       b_ptr, k, n, /*trans_b=*/false,
	       c_ptr, m, n);
    a_ptr += k * m;
    b_ptr += k * n;
    c_ptr += m * n;
  }
}

void GroupedGemmVariableK(torch::Tensor a,
			  torch::Tensor b,
			  torch::Tensor c,
			  torch::Tensor batch_sizes) {
  // We expected a CUDA tensor with two dimensions and shape
  // (tokens, hidden_out) for 'b'.
  TORCH_CHECK(b.is_cuda());
  TORCH_CHECK(b.ndimension() == 2);
  TORCH_CHECK(b.scalar_type() == torch::kBFloat16);

  // Validate the dimensions.
  int64_t tokens = a.size(0), num_experts = batch_sizes.size(0);
  int64_t m = a.size(1), n = b.size(1);

  // Validate that we have the same contraction dimension.
  TORCH_CHECK(tokens == b.size(0));

  // Validate the output shape.
  TORCH_CHECK(c.is_cuda());
  TORCH_CHECK(c.ndimension() == 3);
  TORCH_CHECK(c.scalar_type() == torch::kBFloat16);
  TORCH_CHECK(c.size(0) == num_experts);
  TORCH_CHECK(c.size(1) == m);
  TORCH_CHECK(c.size(2) == n);

  // Run the computation.
  CublasGroupedGemmVariableK(a, b, c, batch_sizes);
}

// NOTE: We only support dynamic group sizes for the 'a' tensor. Tensor 'b' is
// assumed to be batched with fixed sized batches.
//
// TODO(tgale): Validate alignment is true for every batch element.
void GroupedGemm(torch::Tensor a,
		 torch::Tensor b,
		 torch::Tensor c,
		 torch::Tensor batch_sizes,
		 bool trans_a, bool trans_b) {
  // NOTE: We only support 'trans_a' or 'trans_b', not both.
  TORCH_CHECK(!(trans_a && trans_b));

#if !defined(GROUPED_GEMM_CUTLASS)
  // No way to run cuBLAS kernels if the problem dimensions are not known on the host.
  TORCH_CHECK(batch_sizes.is_cpu());
#else
  // CUTLASS can handle both CPU- and CUDA-resident problem dimensions.
  TORCH_CHECK(batch_sizes.is_cuda() || batch_sizes.is_cpu());
#endif
  TORCH_CHECK(batch_sizes.ndimension() == 1);
  TORCH_CHECK(batch_sizes.scalar_type() == torch::kInt64);

  // We expected a CUDA tensor with two dimensions and shape
  // (tokens, hidden_in) for 'a'.
  TORCH_CHECK(a.is_cuda());
  TORCH_CHECK(a.ndimension() == 2);
  TORCH_CHECK(a.scalar_type() == torch::kBFloat16);

#if !defined(GROUPED_GEMM_CUTLASS)
  if (trans_a) {
    // If we can't use CUTLASS for the transposed cases, defer to the variable 'k' helper using cuBLAS
    // for the rest of the op.
    GroupedGemmVariableK(a, b, c, batch_sizes);
    return;
  }
#endif

  TORCH_CHECK(b.is_cuda());
  TORCH_CHECK(c.is_cuda());
  TORCH_CHECK(b.scalar_type() == torch::kBFloat16);
  TORCH_CHECK(c.scalar_type() == torch::kBFloat16);

  // The expected shapes of 'b' and 'c' are:
  //   * when 'trans_a' is set: b=(tokens, hidden_out),                 c=(num_experts, hidden_in, hidden_out)
  //   * when 'trans_b' is set: b=(num_experts, hidden_out, hidden_in), c=(tokens, hidden_out)
  //   * otherwise:             b=(num_experts, hidden_in, hidden_out), c=(tokens, hidden
  size_t hidden_in{}, hidden_out{};
  if (trans_a) {
    hidden_in = a.size(1);
    hidden_out = b.size(1);

    TORCH_CHECK(b.ndimension() == 2);
    TORCH_CHECK(c.ndimension() == 3);
    TORCH_CHECK(b.size(0) == a.size(0));
    TORCH_CHECK(c.size(0) == batch_sizes.size(0));
    TORCH_CHECK(c.size(1) == hidden_in);
    TORCH_CHECK(c.size(2) == hidden_out);
  } else {
    TORCH_CHECK(b.ndimension() == 3);
    TORCH_CHECK(c.ndimension() == 2);

    // Validate the contraction dimensions match.
    int64_t tokens = a.size(0), num_experts = b.size(0);
    hidden_in = trans_b ? b.size(2) : b.size(1);
    hidden_out = trans_b ? b.size(1) : b.size(2);
    TORCH_CHECK(hidden_in == a.size(1));

    // Validate that we have one size per expert.
    TORCH_CHECK(batch_sizes.size(0) == num_experts);
  }

  // NOTE: We support transposition through the 'trans_b' flag.
  TORCH_CHECK(a.is_contiguous());
  TORCH_CHECK(b.is_contiguous());
  TORCH_CHECK(c.is_contiguous());

#if !defined(GROUPED_GEMM_CUTLASS)
  CublasGroupedGemm(a, b, c, batch_sizes, trans_b);
  return;
#else
  // The `coord_template` argument contains `kDynamicDim` as one of its dimensions
  // as a placeholder. This placeholder is later expanded into the actual dimension
  // for every element of the batch,  either on the host or on the device
  // (if we can't do in on the host).
  const auto coord_template = trans_a
    ? cutlass::gemm::GemmCoord(hidden_in, hidden_out, kDynamicDim)
    : cutlass::gemm::GemmCoord(kDynamicDim, hidden_out, hidden_in);
  if (trans_a) {
    CutlassGroupedGemm<true, false>(a, b, c, batch_sizes, coord_template);
    return;
  }
  if (trans_b) {
    CutlassGroupedGemm<false, true>(a, b, c, batch_sizes, coord_template);
    return;
  }
  CutlassGroupedGemm<false, false>(a, b, c, batch_sizes, coord_template);
  return;
#endif
}

}  // namespace grouped_gemm
